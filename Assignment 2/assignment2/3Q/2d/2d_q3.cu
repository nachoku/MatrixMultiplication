#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <math.h>
#include <assert.h>
#include <helper_string.h>  // helper for shared functions common to CUDA Samples

// // CUDA runtime
#include <hip/hip_runtime.h>
#include <hipblas.h>

// // CUDA and CUBLAS functions
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
typedef struct _matrixSize      // Optional Command-line multiplier for matrix sizes
{
    unsigned int uiWA, uiHA, uiWB, uiHB, uiWC, uiHC;
} sMatrixSize;

void randomInit(float *data, int size)
{
    for (int i = 0; i < size; ++i)
    {
        // data[i]=float(i);
        // printf("%f ", data[i]);
        data[i] = rand() / (float)RAND_MAX;
      }
        
    // printf("\n");
}

void
matrixMulCPU(float *C, const float *A, const float *B, unsigned int hA, unsigned int wA, unsigned int wB)
{
    for (unsigned int i = 0; i < hA; ++i)
        for (unsigned int j = 0; j < wB; ++j)
        {
            double sum = 0;

            for (unsigned int k = 0; k < wA; ++k)
            {
                double a = A[i * wA + k];
                double b = B[k * wB + j];
                sum += a * b;
            }

            C[i * wB + j] = (float)sum;
        }
}

void printDiff(float *data1, float *data2, int width, int height, int iListLength, float fListTol)
{
    printf("Listing first %d Differences > %.6f...\n", iListLength, fListTol);
    int i,j,k;
    int error_count=0;

    for (j = 0; j < height; j++)
    {
        if (error_count < iListLength)
        {
            // printf("\n  Row %d:\n", j);
        }

        for (i = 0; i < width; i++)
        {
            k = j * width + i;
            float fDiff = fabs(data1[k] - data2[k]);

            if (fDiff > fListTol)
            {
                if (error_count < iListLength)
                {
                    printf("    Loc(%d,%d)\tCPU=%.5f\tGPU=%.5f\tDiff=%.6f\n", i, j, data1[k], data2[k], fDiff);
                }

                error_count++;
            }
        }
    }

    printf(" \n  Total Errors = %d\n", error_count);
}

__global__
void element(int n, int row_num_b, float *a, float *b, float *c)
{
  int row=blockIdx.x*blockDim.x+threadIdx.x;
  int col=blockIdx.y*blockDim.y+threadIdx.y;

  if(row<row_num_b && col<row_num_b)
  {
    // printf("Thread: %d \n ", i);

    double temp=0;
    for(int j=0;j<row_num_b;j++)
    {
      // printf("quot:%d - rem:%d \n A: %d: %f \n B: %d: %f \n\n", quot, rem, row_num_b*quot + j, a[row_num_b*quot + j], row_num_b*j + rem, b[row_num_b*j + rem]);
      temp+=a[row_num_b*row + j] * b[row_num_b*j + col];
    }
    c[row*row_num_b + col]=(float)temp;
    // printf("%f ", c[quot*row_num_b + rem]);
  }
  
  
}

int main(void)
{
  int nIter = 30;
  //Set Matrix Sizes
  sMatrixSize matrix_size;
  int mul=8;
  matrix_size.uiWA = mul* 160;
  matrix_size.uiHA = mul* 160;
  matrix_size.uiWB = mul* 160;
  matrix_size.uiHB = mul* 160;
  matrix_size.uiWC = mul* 160;
  matrix_size.uiHC = mul* 160;

  printf("MatrixA(%u,%u), MatrixB(%u,%u), MatrixC(%u,%u)\n",
           matrix_size.uiHA, matrix_size.uiWA,
           matrix_size.uiHB, matrix_size.uiWB,
           matrix_size.uiHC, matrix_size.uiWC);

    if( matrix_size.uiWA != matrix_size.uiHB ||
        matrix_size.uiHA != matrix_size.uiHC ||
        matrix_size.uiWB != matrix_size.uiWC)
    {
       printf("ERROR: Matrix sizes do not match!\n");
       exit(-1);
    }
  //Number of Elements
  unsigned int size_A = matrix_size.uiWA * matrix_size.uiHA;
  unsigned int size_B = matrix_size.uiWB * matrix_size.uiHB;
  unsigned int size_C = matrix_size.uiWC * matrix_size.uiHC;
  // printf("Size of A: %d \n", size_A);
  // printf("Size of B: %d \n", size_B);
  //Memory Size
  unsigned int mem_size_A = sizeof(float) * size_A;
  unsigned int mem_size_B = sizeof(float) * size_B;
  unsigned int mem_size_C = sizeof(float) * size_C;

  //Initialize pointer variables
  float *a, *b, *c;

  //Allocate Space for Matrix on Host & Device- Pointer Variables
  hipMallocManaged(&a, mem_size_A);
  hipMallocManaged(&b, mem_size_B);
  hipMallocManaged(&c, mem_size_C);

  //Fill Elements
  randomInit(a, size_A);
  randomInit(b, size_B);  


  unsigned int N=size_C;

  // Perform Matrix Multiplication & Record
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  //Warmup Execution
  hipDeviceProp_t deviceProp;
  int block_size = (deviceProp.major < 2) ? 16 : 32;

  dim3 threads(8, 8);
  dim3 blocks(matrix_size.uiWC / threads.x, matrix_size.uiHC / threads.y);
  element<<<blocks, threads>>>(N, matrix_size.uiHB, a, b, c);

  //Actual execution
  hipEventRecord(start, NULL);
  for (int j = 0; j < nIter; j++)
  {
     element<<<blocks, threads>>>(N, matrix_size.uiHB, a, b, c);
  }
  hipEventRecord(stop, NULL);
  hipEventSynchronize(stop);
  float msecTotal = 0.0f;
  hipEventElapsedTime(&msecTotal, start, stop);

  // Compute and print the performance
  float msecPerMatrixMul = msecTotal / nIter;
  double flopsPerMatrixMul = 2.0 * (double)matrix_size.uiHC * (double)matrix_size.uiWC * (double)matrix_size.uiHB;
  double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
  
  printf( "Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops\n",
      gigaFlops, msecPerMatrixMul, flopsPerMatrixMul);

  //Wait for GPU Finish
  hipDeviceSynchronize();


  float *reference = (float *)malloc(mem_size_C);
  matrixMulCPU(reference, a, b, matrix_size.uiHA, matrix_size.uiWA, matrix_size.uiWB);

bool resCUBLAS = sdkCompareL2fe(reference, c, size_C, 1.0e-6f);

    if (resCUBLAS != true)
    {
  printDiff(reference, c, matrix_size.uiWC, matrix_size.uiHC, 100, 1.0e-4f);
    }

    printf("Comparing CUBLAS Matrix Multiply with CPU results: %s\n", (true == resCUBLAS) ? "PASS" : "FAIL");

  //Release Resources
  // hipFree(d_a);
  // hipFree(d_b);
  // hipFree(d_c);
  
  // free(a);
  // free(b);
  // free(c);
  hipDeviceReset();
}
